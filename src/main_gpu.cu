/*
 * Created by zx on 19-5-15.
 */
#include "gpu/render.cuh"

int main(int argc, char **argv) {
    using namespace utils;
    if (argc != 4)
        return 0;
    printDeviceProperty();
    std::vector<Sphere_GPU> spheres_;
    std::vector<Cube_GPU> cubes_;
    std::vector<Plane_GPU> planes_;
    std::vector<RotaryBezier_GPU> beziers_;
    std::vector<TriangleMeshObject_GPU> meshes_;
    spheres_.emplace_back(
            Sphere_GPU(Vector3(150, 1e5, 181.6), 1e5, Vector3(.9, .9, .9), Vector3(), BRDFs[FLOOR])); //bottom
    spheres_.emplace_back(
            Sphere_GPU(Vector3(50, -1e5 + 381.6, 81.6), 1e5, Vector3(.75, .75, .75), Vector3(), BRDFs[WALL])); // top
    spheres_.emplace_back(
            Sphere_GPU(Vector3(375, 16.5 + 8, 25), 16.5, Vector3(.9, .9, .75) * .999, Vector3(), BRDFs[GLASS]));
    spheres_.emplace_back(Sphere_GPU(Vector3(250, 1181.6 - .9, 81.6), 800, Vector3(), Vector3(50, 50, 50),
                                     BRDFs[LIGHT])); // top light
    Texture_GPU lightcube;
    lightcube.re_idx = 1.3, lightcube.color = Vector3(0.85, 0.85, 0.7), lightcube.emission = Vector3(),
            lightcube.setBRDF(BRDFs[DIFFUSE]);
    cubes_.emplace_back(
            Cube_GPU(Vector3(350, 0, 0), Vector3(400, 8, 50), lightcube));
    planes_.emplace_back(Plane_GPU(Vector3(-1, 0, 0), 1, Vector3(.75, .75, .75), Vector3(), BRDFs[WALL]));  // left
    //planes_.emplace_back(Plane_GPU(Vector3(1, 0, 0), 400, Vector3(.25, .25, .75), Vector3(), DIFF, 1.5)); // right
    planes_.emplace_back(Plane_GPU(Vector3(0, 0, 1), 500, Vector3(.75, .75, .75), Vector3(), BRDFs[WALL]));  // front
    planes_.emplace_back(Plane_GPU(Vector3(0, 0, -1), 0, Vector3(.25, .5, .75), Vector3(), BRDFs[WALL])); // back

    cv::Mat _oilpainting = cv::imread("../texture/oil_painting.png");
    cv::Mat _watercolor = cv::imread("../texture/watercolor.jpg");
    hipTextureObject_t oilpainting = cvMat2CudaTexture(_oilpainting);
    hipTextureObject_t watercolor = cvMat2CudaTexture(_watercolor);
    Texture_GPU oil_painting, watercolor_texture;
    oil_painting.color = Vector3(.75, .75, .75);
    oil_painting.emission = Vector3();
    oil_painting.setBRDF(BRDFs[WALL]);
    oil_painting.img_w = _oilpainting.cols;
    oil_painting.img_h = _oilpainting.rows;
    oil_painting.mapped_image = oilpainting;
    oil_painting.mapped_transform = Transform2D(0, -2 / 450., 2 / 600., 0, 2, 0);
    planes_.emplace_back(Plane_GPU(Vector3(1, 0, 0), 400, oil_painting));
    watercolor_texture.color = Vector3(.9, .9, .5) * .999;
    watercolor_texture.emission = Vector3();
    watercolor_texture.setBRDF(BRDFs[DIFFUSE]);
    watercolor_texture.img_w = _watercolor.cols;
    watercolor_texture.img_h = _watercolor.rows;
    watercolor_texture.mapped_image = watercolor;
    watercolor_texture.mapped_transform = Transform2D(1 / M_PI, 0, 0, .5 / M_PI, 0, 0.25);
    //spheres_.emplace_back(Sphere_GPU(Vector3(280, 13, 103), 13, watercolor_texture));
    spheres_.emplace_back(Sphere_GPU(Vector3(260, 13, 133), 13, Vector3(.75, .75, .75), Vector3(), BRDFs[METAL]));


    double xscale = 2, yscale = 2;
    std::vector<Point2D> ctrl_pnts = {{0. / xscale,  0. / yscale},
                                      {13. / xscale, 0. / yscale},
                                      {30. / xscale, 10. / yscale},
                                      {30. / xscale, 20. / yscale},
                                      {30. / xscale, 30. / yscale},
                                      {25. / xscale, 40. / yscale},
                                      {15. / xscale, 50. / yscale},
                                      {10. / xscale, 70. / yscale},
                                      {20. / xscale, 80. / yscale}};
    Bezier2D cpu_bezier(ctrl_pnts);

    watercolor_texture.setBRDF(BRDFs[CERAMIC]);
    watercolor_texture.mapped_transform = Transform2D(-1., 0, 0, .5 / M_PI, 0, 0.25);
    beziers_.emplace_back(RotaryBezier_GPU(Vector3(310, 3, 205), cpu_bezier.toGPU(), watercolor_texture));

    auto param = loadObject("../model/angel_lucy.obj");
    KDTree cpu_tree(std::get<0>(param), std::get<1>(param), std::get<2>(param));
    KDTree_GPU gpu_tree = cpu_tree.toGPU();
    meshes_.emplace_back(
            TriangleMeshObject_GPU(utils::Vector3(290, .5 - 1.19, 109), 0.1, gpu_tree, Vector3(.75, .75, .75), Vector3(),
                                   BRDFs[DIFFUSE]));

    //debug_kernel<<<1,1>>>(convertToKernel(spheres), convertToKernel(cubes), convertToKernel(planes), convertToKernel(beziers));
    //hipDeviceSynchronize();

    // camera params
    Camera cam = {
            atoi(argv[2]), atoi(argv[3]),
            Vector3(150, 40, 295.6), Vector3(0.4, -0.008612, -0.35).normalize(),
            0.6135, 0.2, 313
    };

    // render
    const dim3 nblocks(cam.w / 16u, cam.h / 16u);
    const dim3 nthreads(16u, 16u);
    KernelArray<utils::Vector3> gpu_out = createKernelArr<utils::Vector3>(static_cast<size_t>(cam.w * cam.h));
    printf("Memory copied to GPU, now start executing render kernel...\n");
    render_wrapper(nblocks, nthreads, makeKernelArr(spheres_), makeKernelArr(cubes_), makeKernelArr(planes_), makeKernelArr(
                                    beziers_), makeKernelArr(meshes_), cam, atoi(argv[1]) / 4, gpu_out);
    std::vector<Vector3> res = makeStdVector(gpu_out);
    releaseKernelArr(gpu_out);
    FILE *f = fopen("image.ppm", "w");
    fprintf(f, "P3\n%d %d\n%d\n", cam.w, cam.h, 255);
    for (int i = 0; i < cam.w * cam.h; ++i)
        fprintf(f, "%d %d %d ", toUInt8(res[i].x()), toUInt8(res[i].y()), toUInt8(res[i].z()));
    fclose(f);
    return 0;
}
