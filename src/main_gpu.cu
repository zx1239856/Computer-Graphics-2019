#include "hip/hip_runtime.h"
/*
 * Created by zx on 19-5-15.
 */
#include "gpu/render.cuh"

int main(int argc, char **argv) {
    using namespace utils;
    if (argc != 4)
        return 0;
    printDeviceProperty();
    std::vector<Sphere_GPU> spheres_;
    std::vector<Cube_GPU> cubes_;
    std::vector<Plane_GPU> planes_;
    std::vector<RotaryBezier_GPU> beziers_;
    std::vector<TriangleMeshObject_GPU> meshes_;
    spheres_.emplace_back(
            Sphere_GPU(Vector3(50, -1e5 + 281.6, 81.6), 1e5, Vector3(.75, .75, .75), Vector3(), BRDFs[WALL])); // top
    spheres_.emplace_back(Sphere_GPU(Vector3(350, 1081.6 - 1.3, 231.6), 800, Vector3(), Vector3(60, 60, 60),
                                     BRDFs[LIGHT])); // top light
    Texture_GPU lightcube;
    lightcube.re_idx = 1.3, lightcube.color = Vector3(.15, .35, .55), lightcube.emission = Vector3(),
            lightcube.setBRDF(BRDFs[DIFFUSE]);
    cubes_.emplace_back(
            Cube_GPU(Vector3(340, 0, 0), Vector3(400, 5, 60), lightcube));
    planes_.emplace_back(Plane_GPU(Vector3(-1, 0, 0), 1, Vector3(.75, .75, .75), Vector3(), BRDFs[WALL]));  // left
    //planes_.emplace_back(Plane_GPU(Vector3(1, 0, 0), 400, Vector3(.25, .25, .75), Vector3(), DIFF, 1.5)); // right
    planes_.emplace_back(Plane_GPU(Vector3(0, 0, 1), 500, Vector3(.75, .75, .75), Vector3(), BRDFs[WALL]));  // front
    
    cv::Mat _oilpainting = cv::imread("../texture/oil_painting.png");
    cv::Mat _watercolor = cv::imread("../texture/watercolor.jpg");
    cv::Mat _floor = cv::imread("../texture/floor.jpg");
    cv::Mat _wall = cv::imread("../texture/wall.jpg");
    hipTextureObject_t oilpainting = cvMat2CudaTexture(_oilpainting);
    hipTextureObject_t watercolor = cvMat2CudaTexture(_watercolor);
    hipTextureObject_t floor = cvMat2CudaTexture(_floor);
    hipTextureObject_t wall = cvMat2CudaTexture(_wall);
    Texture_GPU oil_painting, watercolor_texture, floor_texture, wall_texture;
    floor_texture.color = Vector3(.75, .75, .75);
    floor_texture.emission = Vector3();
    floor_texture.setBRDF(BRDFs[WALL]);
    floor_texture.img_w = _floor.cols;
    floor_texture.img_h = _floor.rows;
    floor_texture.mapped_image = floor;
    floor_texture.mapped_transform = Transform2D(0, -5 / 918., 5 / 1024., 0, 2, 0);
    wall_texture.color = Vector3(.75, .75, .75);
    wall_texture.emission = Vector3();
    wall_texture.setBRDF(BRDFs[WALL]);
    wall_texture.img_w = _wall.cols;
    wall_texture.img_h = _wall.rows;
    wall_texture.mapped_image = wall;
    wall_texture.mapped_transform = Transform2D(0, -2 / 1350., 2 / 2400., 0, 0, 0);
    oil_painting.color = Vector3(.75, .75, .75);
    oil_painting.emission = Vector3();
    oil_painting.setBRDF(BRDFs[WALL]);
    oil_painting.img_w = _oilpainting.cols;
    oil_painting.img_h = _oilpainting.rows;
    oil_painting.mapped_image = oilpainting;
    oil_painting.mapped_transform = Transform2D(0, -2 / 450., 2 / 600., 0, 2, 0);
    planes_.emplace_back(Plane_GPU(Vector3(1, 0, 0), 400, oil_painting));
    watercolor_texture.color = Vector3(.9, .9, .5) * .999;
    watercolor_texture.emission = Vector3();
    watercolor_texture.setBRDF(BRDFs[DIFFUSE]);
    watercolor_texture.img_w = _watercolor.cols;
    watercolor_texture.img_h = _watercolor.rows;
    watercolor_texture.mapped_image = watercolor;
    watercolor_texture.mapped_transform = Transform2D(1 / M_PI, 0, 0, .5 / M_PI, 0, 0.25);
    //spheres_.emplace_back(Sphere_GPU(Vector3(280, 13, 103), 13, watercolor_texture));
    spheres_.emplace_back(Sphere_GPU(Vector3(265, 13, 100), 13, Vector3(.75, .75, .75), Vector3(), BRDFs[METAL]));
    spheres_.emplace_back(Sphere_GPU(Vector3(300, 10, 200), 10, Vector3(.75, .9, .9), Vector3(), BRDFs[GLASS]));
    spheres_.emplace_back(Sphere_GPU(Vector3(280, 8, 135), 8, Vector3(.75, .9, .65), Vector3(), BRDFs[GLASS]));
    spheres_.emplace_back(Sphere_GPU(Vector3(270, 5, 155), 5, Vector3(.75, .75, .35), Vector3(), BRDFs[GLASS]));
    planes_.emplace_back(Plane_GPU(Vector3(0, 0, -1), 0, wall_texture)); // back
    planes_.emplace_back(Plane_GPU(Vector3(0, 1, 0), 0, floor_texture)); // bottom
    double xscale = 1.5, yscale = 1.5;
    std::vector<Point2D> ctrl_pnts = {{0. / xscale,  0. / yscale},
                                      {13. / xscale, 0. / yscale},
                                      {30. / xscale, 10. / yscale},
                                      {30. / xscale, 20. / yscale},
                                      {30. / xscale, 30. / yscale},
                                      {25. / xscale, 40. / yscale},
                                      {15. / xscale, 50. / yscale},
                                      {10. / xscale, 70. / yscale},
                                      {20. / xscale, 80. / yscale}};
    Bezier2D cpu_bezier(ctrl_pnts);

    watercolor_texture.mapped_transform = Transform2D(-1., 0, 0, .5 / M_PI, 0, 0.25);
    beziers_.emplace_back(RotaryBezier_GPU(Vector3(370, 5.5, 30), cpu_bezier.toGPU(), watercolor_texture));

    auto param = loadObject("../model/angel_lucy.obj");
    KDTree cpu_tree(std::get<0>(param), std::get<1>(param), std::get<2>(param));
    KDTree_GPU gpu_tree = cpu_tree.toGPU();
    meshes_.emplace_back(
            TriangleMeshObject_GPU(utils::Vector3(345, .5 - 1.19, 169), 0.1, gpu_tree, Vector3(.75, .75, .75), Vector3(),
                                   BRDFs[DIFFUSE]));

    //debug_kernel<<<1,1>>>(convertToKernel(spheres), convertToKernel(cubes), convertToKernel(planes), convertToKernel(beziers));
    //hipDeviceSynchronize();

    // camera params
    Camera cam = {
            atoi(argv[2]), atoi(argv[3]),
            Vector3(150, 40, 295.6), Vector3(0.4, -0.008612, -0.35).normalize(),
            0.5535, 3.3, 223
    };

    // render
    const dim3 nblocks(cam.w / 16u, cam.h / 16u);
    const dim3 nthreads(16u, 16u);
    KernelArray<utils::Vector3> gpu_out = createKernelArr<utils::Vector3>(static_cast<size_t>(cam.w * cam.h));
    printf("Memory copied to GPU, now start executing render kernel...\n");
    render_wrapper(nblocks, nthreads, makeKernelArr(spheres_), makeKernelArr(cubes_), makeKernelArr(planes_), makeKernelArr(
                                    beziers_), makeKernelArr(meshes_), cam, atoi(argv[1]) / 4, gpu_out);
    std::vector<Vector3> res = makeStdVector(gpu_out);
    releaseKernelArr(gpu_out);
    FILE *f = fopen("image.ppm", "w");
    fprintf(f, "P3\n%d %d\n%d\n", cam.w, cam.h, 255);
    for (int i = 0; i < cam.w * cam.h; ++i)
        fprintf(f, "%d %d %d ", toUInt8(res[i].x()), toUInt8(res[i].y()), toUInt8(res[i].z()));
    fclose(f);
    return 0;
}
